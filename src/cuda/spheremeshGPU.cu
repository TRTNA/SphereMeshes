#include "hip/hip_runtime.h"
#include <cuda/spheremeshGPU.h>
#include <spheremeshes/point.h>
#include <spheremeshes/spheremesh.h>
#include <spheremeshes/sphere.h>
#include <spheremeshes/capsuloid.h>

#include <utils/random.h>
#include <utils/aabb.h>

#include <glm/glm.hpp>

#include <stdio.h>

#include <cmath>

#include <array>

#include <hiprand/hiprand_kernel.h>

using glm::vec3;
using std::array;

typedef unsigned long ulong;

__device__ const float GPU_EPSILON = 0.0001f;

float computeTime(hipEvent_t &e1, hipEvent_t &e2)
{
    float time;
    hipEventElapsedTime(&time, e1, e2);
    return time;
}

void checkError(hipError_t error)
{

    if (error != 0)
    {
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);
        fprintf(stderr, "code: %d, reason: %s\n", error,
                hipGetErrorString(error));
    }
}

#define CHECK(call)                                                \
    {                                                              \
        const hipError_t error = call;                            \
        if (error != hipSuccess)                                  \
        {                                                          \
            fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
            fprintf(stderr, "code: %d, reason: %s\n", error,       \
                    hipGetErrorString(error));                    \
        }                                                          \
    }



__global__ void checkDimensionality(int *dimensionalities, uint chunkSize)
{
    const long tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= chunkSize)
        return;
    if (dimensionalities[tid] == -1)
        dimensionalities[tid] = -2;
}

__global__ void generateRandomPointsInsideSphere(vec3 sphereCenter, float sphereRadius, glm::vec3 *pointsPositions, hiprandState_t *states, uint chunkSize)
{
    const long tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid >= chunkSize)
        return;
    hiprand_init(tid, 0, 0, &states[tid]);

    // random 3D direction
    // NOTE: 2 *n - 1 shifts the interval from 0, 1 to -1, 1
    vec3 direction = glm::normalize(vec3(2.f * hiprand_uniform(&states[tid]) - 1.f, 2.f * hiprand_uniform(&states[tid]) - 1.f, 2.f * hiprand_uniform(&states[tid]) - 1.f));

    // hiprand_uniform return number in (0,1], multiplied by radius return a value in (0, radius] so inside the sphere
    float extent = hiprand_uniform(&states[tid]) * sphereRadius;

    pointsPositions[tid] = vec3(sphereCenter + direction * extent);
}

__global__ void pushOutsideSphere(glm::vec3 *positions, glm::vec3 *normals, int *dimensionalities, Sphere sphere, uint chunkSize)
{
    const long tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= chunkSize)
        return;

    int dimensionality = dimensionalities[tid];

    if (dimensionality == -2)
        return;

    glm::vec3 pos = positions[tid];
    glm::vec3 CtoPos = pos - sphere.center;
    const float CtoPossqrd = glm::dot(CtoPos, CtoPos);

    // pos is outside sphere
    if (CtoPossqrd > sphere.radius * sphere.radius - GPU_EPSILON)
    {
        // se il punto aveva dimensionalità -1, non dire nulla perché non sai se è interno a un'altra primitiva, ma solo che è esterno a questa
        // se il punto aveva dimensionalità != -1, non fare nulla perchè serve mantenerne l'informazione
        return;
    }

    // if we are here, pos is inside the sphere
    dimensionalities[tid] = 0;
    CtoPos = glm::normalize(CtoPos);
    positions[tid] = sphere.center + sphere.radius * CtoPos;
    normals[tid] = CtoPos;
}

__device__ void pushOutsideCapsuloid(int tid, glm::vec3 *positions, glm::vec3 *normals, int *dimensionalities, Sphere &A, Sphere &B, float factor, glm::vec3 &BminusA)
{
    vec3 pos = positions[tid];

    float k = glm::dot(pos - A.center, BminusA) / glm::dot(BminusA, BminusA);
    vec3 fakeC = A.center + k * BminusA;
    float d = length(fakeC - pos);

    k += (factor * d);

    const float clampedK = glm::clamp(k, 0.0f, 1.0f);

    const vec3 C = A.center + clampedK * BminusA;

    const vec3 CtoPos = pos - C;

    const float CtoPossqrd = glm::dot(CtoPos, CtoPos);

    const float interpRadius = A.radius * (1.0f - clampedK) + B.radius * clampedK;

    // pos is outside the capsule, dimensionality is -1 (not pushed out)
    if (CtoPossqrd > interpRadius * interpRadius - GPU_EPSILON)
    {
        // se il punto aveva dimensionalità -1, non dire nulla perché non sai se è interno a un'altra primitiva, ma solo che è esterno a questa
        // se il punto aveva dimensionalità != -1, non fare nulla perchè serve mantenerne l'informazione
        return;
    }

    // if we are here, pos is inside the capsule
    // dimensionality depends on K value
    // if clampedK == k then pos is inside the cylinder, so dimensionality = 1
    // else pos is inside one of the spheres, so dimensionality = 0
    bool dim = k == clampedK;
    dimensionalities[tid] = (int)dim;
    const vec3 normal = glm::normalize(CtoPos);

    positions[tid] = C + interpRadius * normal;
    normals[tid] = normal;
}

__global__ void pushOutsideCapsuloidKernel(glm::vec3 *positions, glm::vec3 *normals, int *dimensionalities, Sphere A, Sphere B, float factor, glm::vec3 BminusA, uint chunkSize)
{
    const long tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= chunkSize)
        return;

    int dimensionality = dimensionalities[tid];

    if (dimensionality == -2)
        return;

    pushOutsideCapsuloid(tid, positions, normals, dimensionalities, A, B, factor, BminusA);
}

__global__ void pushOutsideSphereTriangle(glm::vec3 *positions, glm::vec3 *normals, int *dimensionalities, Sphere s0, Sphere s1, Sphere s2, glm::mat4 upperProjMatrix, glm::mat4 lowerProjMatrix, glm::vec3 planeN, uint chunkSize)
{
    const long tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= chunkSize)
        return;

    int dimensionality = dimensionalities[tid];
    if (dimensionality == -2)
        return;

    glm::vec3 pos = positions[tid];

    const vec3 q = pos - s0.center;
    float d, a, b, c;
    const glm::mat3 projMatrix = glm::dot(q, planeN) < 0 ? lowerProjMatrix : upperProjMatrix;
    const vec3 res = projMatrix * q;
    d = res.z;
    a = res.x;
    b = res.y;
    c = (1.0f - a - b);

    if (b < 0.0f)
    {
        // PUSH OUTSIDE CAPSULE V0V1
        glm::vec3 s1minuss0 = s1.center - s0.center;
        float factor = (s1.radius - s0.radius) / glm::dot(s1minuss0, s1minuss0);
        pushOutsideCapsuloid(tid, positions, normals, dimensionalities, s0, s1, factor, s1minuss0);
        return;
    }
    if (c < 0.0f)
    {
        // PUSH OUTSIDE CAPSULE V1V2
        glm::vec3 s2minuss1 = s2.center - s1.center;
        float factor = (s2.radius - s1.radius) / glm::dot(s2minuss1, s2minuss1);
        pushOutsideCapsuloid(tid, positions, normals, dimensionalities, s1, s2, factor, s2minuss1);
        return;
    }
    if (a < 0.0f)
    {
        // PUSH OUTSIDE CAPSULE V0V2
        glm::vec3 s2minuss0 = s2.center - s0.center;
        float factor = (s2.radius - s0.radius) / glm::dot(s2minuss0, s2minuss0);
        pushOutsideCapsuloid(tid, positions, normals, dimensionalities, s0, s2, factor, s2minuss0);
        return;
    }

    // PUSH OUTSIDE TRIANGLE
    if (a < 1.0f && b < 1.0f && c < 1.0f)
    {
        vec3 C = c * s0.center + a * s1.center + b * s2.center;
        float interpRadius = c * s0.radius + a * s1.radius + b * s2.radius;
        vec3 CtoPos = pos - C;
        if (d > interpRadius - GPU_EPSILON)
        {
            // se il punto aveva dimensionalità -1, non dire nulla perché non sai se è interno a un'altra primitiva, ma solo che è esterno a questa
            // se il punto aveva dimensionalità != -1, non fare nulla perchè serve mantenerne l'informazione
            return;
        }
        dimensionalities[tid] = 2;
        glm::vec3 normal = glm::normalize(CtoPos);
        positions[tid] = C + interpRadius * normal;
        normals[tid] = normal;
    }
}

void createSphereMeshGPU(SphereMesh &sphereMesh, uint numberOfPoints, std::vector<DimensionalityPoint> &outPoints)
{
    printf("Starting...\n");
    hipSetDevice(0);
    array<hipEvent_t, 11> events;

    for (hipEvent_t &event : events)
    {
        CHECK(hipEventCreate(&event));
    }

    // # 1. Inizializzazione memoria host
    CHECK(hipEventRecord(events[0], 0));
    printf("Inizializzazione memoria host...\n");
    ulong coordinatesBytes = numberOfPoints * sizeof(float) * 3;
    ulong dimensionalityBytes = numberOfPoints * sizeof(int);

    glm::vec3 *hostPositions = (glm::vec3 *)malloc(coordinatesBytes);
    glm::vec3 *hostNormals = (glm::vec3 *)malloc(coordinatesBytes);

    int *tempDimensionalities = (int *)malloc(dimensionalityBytes);

    hiprandState *devStates;
    hipMalloc((void **)&devStates, numberOfPoints * sizeof(hiprandState));

    CHECK(hipEventRecord(events[1], 0));
    // wait until the stop event completes
    CHECK(hipEventSynchronize(events[1]));
    printf("Allocati %lu bytes in memoria host in %f millisecondi...\n", coordinatesBytes + dimensionalityBytes, computeTime(events[0], events[1]));

    // # 2. Inizializzazione memoria device
    printf("Inizializzazione memoria device...\n");

    int *deviceDimensionalities;
    glm::vec3 *devicePositions, *deviceNormals;
    CHECK(hipEventRecord(events[2]));

    CHECK(hipMalloc((void **)&devicePositions, coordinatesBytes));
    CHECK(hipMalloc((void **)&deviceNormals, coordinatesBytes));

    CHECK(hipMalloc((void **)&deviceDimensionalities, dimensionalityBytes));
    CHECK(hipMemset(deviceDimensionalities, -1, numberOfPoints * sizeof(int)));

    CHECK(hipEventRecord(events[3]));
    CHECK(hipEventSynchronize(events[3]));

    printf("Allocati %lu bytes in memoria device in %f millisecondi...\n", coordinatesBytes + dimensionalityBytes, computeTime(events[2], events[3]));

    // # 3. creazione punti random dentro la bounding sphere della sphere mesh

    printf("Generazione posizioni random su GPU...\n");
    hipStream_t streams[10];
    for (size_t i = 0; i < 10; i++)
    {
        hipStreamCreate(&streams[i]);
    }
    int subBlockSize = 256;
    uint chunkSize = (numberOfPoints / 10) + 1;
    dim3 subGrid((chunkSize / subBlockSize) + 1);
    dim3 subBlock(subBlockSize);

    float bsRadius = sphereMesh.boundingSphere.radius;
    vec3 bsCenter = sphereMesh.boundingSphere.center;

    CHECK(hipEventRecord(events[4]));
    for (size_t i = 0; i < 10; i++)
    {
        generateRandomPointsInsideSphere<<<subGrid, subBlock, 0, streams[i]>>>(bsCenter, bsRadius, &devicePositions[chunkSize * i], &devStates[chunkSize * i], chunkSize);
    }
    CHECK(hipDeviceSynchronize());
    checkError(hipGetLastError());
    CHECK(hipEventRecord(events[5]));
    CHECK(hipEventSynchronize(events[5]));
    printf("Generate %u posizioni in memoria device in %f millisecondi...\n", numberOfPoints, computeTime(events[4], events[5]));

    // # 4. Creazione contesto loop
    const uint maxTries = 5U;

    CHECK(hipEventRecord(events[6]));

    const uint singletonStart = 0;
    const uint edgeStart = singletonStart + sphereMesh.singletons.size();
    const uint triangleStart = edgeStart + sphereMesh.capsuloids.size();
    const uint maxUniqueIdx = triangleStart + sphereMesh.sphereTriangles.size();

    // # 5. Loop di creazione dei punti
    for (size_t i = 0; i < 10; i++)
    {
        for (uint tries = 0; tries < maxTries; tries++)
        {
            // Primitives loop
            for (size_t uniqueIdx = 0; uniqueIdx < maxUniqueIdx; uniqueIdx++)
            {
                if (uniqueIdx >= singletonStart && uniqueIdx < edgeStart)
                {
                    pushOutsideSphere<<<subGrid, subBlock, 0, streams[i]>>>(&devicePositions[chunkSize * i], &deviceNormals[chunkSize * i], &deviceDimensionalities[chunkSize * i], sphereMesh.spheres.at(sphereMesh.singletons.at(uniqueIdx)), chunkSize);
                }
                else if (uniqueIdx >= edgeStart && uniqueIdx < triangleStart)
                {
                    Capsuloid &caps = sphereMesh.capsuloids.at(uniqueIdx - edgeStart);

                    pushOutsideCapsuloidKernel<<<subGrid, subBlock, 0, streams[i]>>>(&devicePositions[chunkSize * i], &deviceNormals[chunkSize * i], &deviceDimensionalities[chunkSize * i], sphereMesh.spheres.at(caps.s0), sphereMesh.spheres.at(caps.s1), caps.factor, caps.S0toS1, chunkSize);
                }
                else if (uniqueIdx >= triangleStart)
                {
                    SphereTriangle &st = sphereMesh.sphereTriangles.at(uniqueIdx - triangleStart);
                    pushOutsideSphereTriangle<<<subGrid, subBlock, 0, streams[i]>>>(&devicePositions[chunkSize * i], &deviceNormals[chunkSize * i], &deviceDimensionalities[chunkSize * i], sphereMesh.spheres.at(st.vertices[0]), sphereMesh.spheres.at(st.vertices[1]), sphereMesh.spheres.at(st.vertices[2]), st.upperProjMatrix, st.lowerProjMatrix, st.planeN, chunkSize);
                }
            }
            checkError(hipGetLastError());
            if (tries == 0)
            {
                // è il primo tentativo, se un punto è rimasto a -1 allora è esterno, va scartato
                //  chiamata a kernel che setta i punti a -1 su -2 (verrano ignorati negli altri kernel)
                checkDimensionality<<<subGrid, subBlock, 0, streams[i]>>>(&deviceDimensionalities[chunkSize * i], chunkSize);
                checkError(hipGetLastError());
            }
        }
    }

    CHECK(hipDeviceSynchronize());

    // # Esce dal ciclo quanto tutti i punti sono o esterni alla sphere mesh o spinti sulla superficie
    CHECK(hipEventRecord(events[7]));
    CHECK(hipEventSynchronize(events[7]));

    printf("Creazione punti TERMINATA in %f millisecondi\n", computeTime(events[6], events[7]));

    // # 6. Copia da memoria device a memoria host (funziona così?)
    printf("Copia dati da device a host...\n");
    CHECK(hipEventRecord(events[8]));
    for (size_t i = 0; i < 9; i++)
    {
        CHECK(hipMemcpyAsync(&hostPositions[chunkSize * i], &devicePositions[chunkSize * i], chunkSize * sizeof(vec3), hipMemcpyDeviceToHost, streams[i]));
        CHECK(hipMemcpyAsync(&hostNormals[chunkSize * i], &deviceNormals[chunkSize * i], chunkSize * sizeof(vec3), hipMemcpyDeviceToHost, streams[i]));
        CHECK(hipMemcpyAsync(&tempDimensionalities[chunkSize * i], &deviceDimensionalities[chunkSize * i], chunkSize * sizeof(int), hipMemcpyDeviceToHost, streams[i]));
    }
    uint cumChunkSize = chunkSize * 9;
    uint lastChunk = numberOfPoints - cumChunkSize;
    CHECK(hipMemcpyAsync(&hostPositions[cumChunkSize], &devicePositions[cumChunkSize], lastChunk * sizeof(vec3), hipMemcpyDeviceToHost, streams[9]));
    CHECK(hipMemcpyAsync(&hostNormals[cumChunkSize], &deviceNormals[cumChunkSize], lastChunk * sizeof(vec3), hipMemcpyDeviceToHost, streams[9]));
    CHECK(hipMemcpyAsync(&tempDimensionalities[cumChunkSize], &deviceDimensionalities[cumChunkSize], lastChunk * sizeof(int), hipMemcpyDeviceToHost, streams[9]));

    CHECK(hipDeviceSynchronize());
    CHECK(hipEventRecord(events[9]));
    CHECK(hipEventSynchronize(events[9]));

    printf("Copia dati da device a host TERMINATA in %f millisecondi\n", computeTime(events[8], events[9]));

    // # 7. Eliminazione memoria allocata sul device
    CHECK(hipFree(devicePositions));
    CHECK(hipFree(deviceNormals));
    CHECK(hipFree(devStates));
    CHECK(hipFree(deviceDimensionalities));
    for (size_t i = 0; i < 10; i++) {
        CHECK(hipStreamDestroy(streams[i]));
    }

    // # 8. Scarto dei punti che non sono stati spinti sulla superficie della sphere mesh (dimensionality != -1)
    // # ovvero punti esterni alla sphere mesh o interni che non sono stati spinti fuori
    outPoints.clear();
    for (size_t i = 0; i < numberOfPoints; i++)
    {
        if (tempDimensionalities[i] == -2)
            continue;
        outPoints.emplace_back(hostPositions[i], hostNormals[i], tempDimensionalities[i]);
    }

    printf("Sono stati ottenuti %zu punti sui %zu richiesti\n", outPoints.size(), numberOfPoints);

    // # 9. TODO: Controllo di essere arrivato al numero di punti desiderato
    // # se non ci sono arrivato, riavvio creazione punti con un certo numero da definire (metà? Tenendo conto del numero di punti scartati?)
    // # per questo punto servirà refactoring profondo del ciclo do/while che andrà inserito in una funzione dedicata (con anche allocazione e distruzione memoria)

    CHECK(hipEventRecord(events[10], 0));
    CHECK(hipEventSynchronize(events[10]));

    printf("L'esecuzione dell'algoritmo (compresa la gestione della memoria) e' durata %f millisecondi\n", computeTime(events[0], events[10]));

    for (auto &event : events)
    {
        CHECK(hipEventDestroy(event));
    }

    // # 10. Eliminazione memoria allocata su host
    delete[] hostPositions;
    delete[] hostNormals;
    delete[] tempDimensionalities;
}