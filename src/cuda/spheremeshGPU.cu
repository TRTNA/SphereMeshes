#include "hip/hip_runtime.h"
#include <cuda/spheremeshGPU.h>
#include <spheremeshes/point.h>

#include <stdio.h>

#include <array>

using std::array;

typedef unsigned long ulong;

float computeTime(hipEvent_t &e1, hipEvent_t &e2)
{
    float time;
    hipEventElapsedTime(&time, e1, e2);
    return time;
}

void checkError(hipError_t error)
{

    if (error != 0)
    {
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);
        fprintf(stderr, "code: %d, reason: %s\n", error,
                hipGetErrorString(error));
    }
}

#define CHECK(call)                                                \
    {                                                              \
        const hipError_t error = call;                            \
        if (error != hipSuccess)                                  \
        {                                                          \
            fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
            fprintf(stderr, "code: %d, reason: %s\n", error,       \
                    hipGetErrorString(error));                    \
        }                                                          \
    }

__global__ void testKernel(float *dX)
{
    dX[blockIdx.x * blockDim.x + threadIdx.x] = 0.0f;
}

void createSphereMeshGPU(SphereMesh &sphereMesh, uint numberOfPoints, std::vector<DimensionalityPoint> &outPoints)
{
    printf("Starting...\n");
    hipSetDevice(0);
    array<hipEvent_t, 11> events;

    for (hipEvent_t &event : events)
    {
        CHECK(hipEventCreate(&event));
    }

    // # 1. Inizializzazione memoria host
    CHECK(hipEventRecord(events[0], 0));
    printf("Inizializzazione memoria host...\n");
    ulong pointsCoordBytes = numberOfPoints * sizeof(float);
    ulong dimensionalityBytes = numberOfPoints * sizeof(int);

    float *hostX = (float *)malloc(pointsCoordBytes);
    float *hostY = (float *)malloc(pointsCoordBytes);
    float *hostZ = (float *)malloc(pointsCoordBytes);
    int *hostDimensionality = (int *)malloc(dimensionalityBytes);
    for (size_t i = 0; i < numberOfPoints; i++)
    {
        hostDimensionality[i] = -1;
    }
    CHECK(hipEventRecord(events[1], 0));
    // wait until the stop event completes
    CHECK(hipEventSynchronize(events[1]));
    printf("Allocati %lu bytes in memoria host in %f millisecondi...\n", pointsCoordBytes * 3 + dimensionalityBytes, computeTime(events[0], events[1]));

    // # 2. TODO inizializzare i punti con valori random dentro la bounding sphere della sphere mesh
    // # on GPU?

    // # 3. Inizializzazione memoria device
    printf("Inizializzazione memoria device...\n");
    float *deviceX, *deviceY, *deviceZ;
    int *deviceDimensionality;
    CHECK(hipEventRecord(events[2]));

    CHECK(hipMalloc((void **)&deviceX, pointsCoordBytes));

    CHECK(hipMalloc((void **)&deviceY, pointsCoordBytes));
    CHECK(hipMalloc((void **)&deviceZ, pointsCoordBytes));
    CHECK(hipMalloc((void **)&deviceDimensionality, dimensionalityBytes));

    CHECK(hipEventRecord(events[3]));
    CHECK(hipEventSynchronize(events[3]));

    printf("Allocati %lu bytes in memoria device in %f millisecondi...\n", pointsCoordBytes * 3 + dimensionalityBytes, computeTime(events[2], events[3]));

    // # 4. Copia da memoria host a memoria device
    printf("Copia dati da host a device...\n");
    CHECK(hipEventRecord(events[4]));
    CHECK(hipMemcpy(deviceX, hostX, pointsCoordBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(deviceY, hostY, pointsCoordBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(deviceZ, hostZ, pointsCoordBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(deviceDimensionality, hostDimensionality, dimensionalityBytes, hipMemcpyHostToDevice));

    CHECK(hipEventRecord(events[5]));
    CHECK(hipEventSynchronize(events[5]));

    printf("Copia dati da host a device TERMINATA in %f millisecondi\n", computeTime(events[4], events[5]));

    // # 6. Creazione contesto loop
    const uint maxTries = 10U;
    uint tries = 0U;
    bool *allNegativeDim;
    CHECK(hipMallocManaged(&allNegativeDim, sizeof(int)));

    CHECK(hipEventRecord(events[6]));

    //TODO tbd wrt points number
    dim3 grid(1);
    dim3 block(32, 32);

    // # 7. Loop di creazione dei punti
    do
    {
        printf("Tentativo %u...", tries);
        // # 7.1 TODO Chiamata al kernel di push outside
        // # pushOutside<<<grid, block>>>(devicePoints, numberOfPoints);

        // # 7.2 Sincronizzazione sul lavoro del kernel pushOutside
        printf("Attesa terminazione kernel...\n");

        testKernel<<<grid, block>>>(deviceX);
        checkError(hipGetLastError());
        CHECK(hipDeviceSynchronize());

        // # 7.3 TODO Chiamata al kernel che controlla se sono tutte negative le dimensionality
        printf("Controllo dimensionalita' punti...\n");
        // # checkAllNegativeDimensionalities<<<grid, block>>>(devicePoints, numberOfPoints, allNegativeDim);
        tries++;
    } while (!(*allNegativeDim) && tries < maxTries);
    // # Esce dal ciclo quanto tutti i punti sono o esterni alla sphere mesh o spinti sulla superficie
    CHECK(hipEventRecord(events[7]));
    CHECK(hipEventSynchronize(events[7]));

    printf("Creazione punti TERMINATA in %f millisecondi\n", computeTime(events[6], events[7]));
    printf("Creazione punti terminata perche' %s\n", tries == maxTries ? "sono stati esauriti i tentativi" : "i punti sono tutti esterni o sulla superficie");

    // # 8. Copia da memoria device a memoria host (funziona così?)
    printf("Copia dati da device a host...\n");
    CHECK(hipEventRecord(events[8]));
    CHECK(hipMemcpy(hostX, deviceX, pointsCoordBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(hostY, deviceY, pointsCoordBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(hostZ, deviceZ, pointsCoordBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(hostDimensionality, deviceDimensionality, dimensionalityBytes, hipMemcpyDeviceToHost));
    CHECK(hipEventRecord(events[9]));
    CHECK(hipEventSynchronize(events[9]));

    printf("Copia dati da device a host TERMINATA in %f millisecondi\n", computeTime(events[8], events[9]));

    // # 9. Eliminazione memoria allocata sul device (memcpy è bloccante, sono sicuro che non mi serva più quando arrivo qui)
    CHECK(hipFree(deviceX));
    CHECK(hipFree(deviceY));
    CHECK(hipFree(deviceZ));
    CHECK(hipFree(deviceDimensionality));

    // # 10. Scarto dei punti che non sono stati spinti sulla superficie della sphere mesh (dimensionality != -1)
    // # ovvero punti esterni alla sphere mesh o interni che non sono stati spinti fuori
    vector<DimensionalityPoint> points;
    for (size_t i = 0; i < numberOfPoints; i++)
    {
        if (hostDimensionality[i] != -1)
            continue;
        points.emplace_back(
            hostX[i],
            hostY[i],
            hostZ[i]);
    }

    printf("Sono stati ottenuti %zu punti sui %zu richiesti\n", points.size(), numberOfPoints);

    // # 11. TODO: Controllo di essere arrivato al numero di punti desiderato
    // # se non ci sono arrivato, riavvio creazione punti con un certo numero da definire (metà? Tenendo conto del numero di punti scartati?)
    // # per questo punto servirà refactoring profondo del ciclo do/while che andrà inserito in una funzione dedicata (con anche allocazione e distruzione memoria)

    // # 12. Eliminazione memoria allocata su host
    delete hostX;
    delete hostY;
    delete hostZ;
    delete hostDimensionality;

    CHECK(hipEventRecord(events[10], 0));
    CHECK(hipEventSynchronize(events[10]));

    printf("L'esecuzione dell'algoritmo (compresa la gestione della memoria) e' durata %f millisecondi\n", computeTime(events[0], events[10]));

    for (auto &event : events)
    {
        CHECK(hipEventDestroy(event));
    }
}